#include "hip/hip_runtime.h"
#include "cuda/cuda.hpp"
#include "utils/SpacialInformations.hpp"
#include "logic/Quadrant.hpp"

void apply_accel(std::vector<std::shared_ptr<Star>> &starList)
{
	for (auto &it: starList) {
		it->setX(it->getX() + it->getAccx());
		it->setY(it->getY() + it->getAccy());
	}
}

void compute_accel_cuda(SpacialInformations *vec, Quadrant *quadrant, float *res, size_t vecSize, size_t quadrantSize) {
	__compute_accel_cuda<<<1, 256>>>(vec, quadrant, res, vecSize, quadrantSize);
}

__global__ void __compute_accel_cuda(SpacialInformations *infoVec, Quadrant *quadrantVec, float *res, size_t infoVecSize, size_t quadrantVecSize)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int gid = blockDim.x * gridDim.x;
	for (; id < infoVecSize; id += gid) {
		for (int j = 0; j < quadrantVecSize; j++) {
			auto d = quadrantVec[j]._width;
			auto r = sqrt(pow(infoVec[id]._x - quadrantVec[j]._cmx, 2) +
				      pow(infoVec[id]._y - quadrantVec[j]._cmy, 2) +
				      SOFTENER);
			if (d / r <= THETA) {
				auto k = quadrantVec[j]._mass * G / (pow(r + 5, 3));
				res[id * 2] = k * (quadrantVec[j]._cmx - infoVec[id]._x);
				res[id * 2 + 1] = k * (quadrantVec[j]._cmx - infoVec[id]._y);
			}
			
		}
	}
}
